/******************************************************************************
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "WeightedSsePostOutputLayer.hpp"
#include "../helpers/getRawPointer.cuh"

#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>


namespace internal {
namespace {


    struct ComputeWeightedSseFn
    {
        int layerSize;

        const char *patTypes;
        real_t *targets;
        real_t *outputs;

        __host__ __device__ real_t operator() (int index) const
        {
            // unpack the tuple
            real_t target = targets[index * 2];
            real_t output = outputs[index];
            real_t weight = targets[index * 2 + 1];

            // check if we have to skip this value
            int patIdx = index / layerSize;
            if (patTypes[patIdx] == PATTYPE_NONE)
                return 0;

            // calculate the error
            real_t diff = (output - target) * weight;
            return (diff * diff);
        }
    };

    struct ComputeOutputErrorFn
    {
        int layerSize;

        const char *patTypes;
        real_t *targets;
        real_t *outputs;

        __host__ __device__ real_t operator() (int index) const
        {
            // unpack the tuple
            real_t target = targets[index * 2];
            real_t output = outputs[index];
            real_t weight = targets[index * 2 + 1];

            // calculate the pattern index
            int patIdx = index / layerSize;

            // check if the pattern is a dummy
            if (patTypes[patIdx] == PATTYPE_NONE)
                return 0;

            // calculate the error
            real_t error = (output - target) * weight;

            return error;
        }
    };
    
} // anonymous namespace
} // namespace anonymous


namespace layers {

    template <typename TDevice>
    WeightedSsePostOutputLayer<TDevice>::WeightedSsePostOutputLayer(const helpers::JsonValue &layerChild, Layer<TDevice> &precedingLayer)
        : PostOutputLayer<TDevice>  (layerChild, precedingLayer, precedingLayer.size() * 2)
    {
    }

    template <typename TDevice>
    WeightedSsePostOutputLayer<TDevice>::~WeightedSsePostOutputLayer()
    {
    }

    template <typename TDevice>
    const std::string& WeightedSsePostOutputLayer<TDevice>::type() const
    {
        static const std::string s("weightedsse");
        return s;
    }

    template <typename TDevice>
    real_t WeightedSsePostOutputLayer<TDevice>::calculateError()
    {
        internal::ComputeWeightedSseFn fn;
        fn.layerSize = this->size() / 2;
        fn.patTypes  = helpers::getRawPointer(this->patTypes());
        fn.targets = helpers::getRawPointer(this->_targets());
        fn.outputs = helpers::getRawPointer(this->_actualOutputs());

        int n = this->curMaxSeqLength() * this->parallelSequences() * this->size() / 2;

        real_t mse = (real_t)0.5 * thrust::transform_reduce(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(0) + n,
            fn,
            (real_t)0,
            thrust::plus<real_t>()
            );

        return mse;
    }

    template <typename TDevice>
    void WeightedSsePostOutputLayer<TDevice>::computeForwardPass()
    {
    }

    template <typename TDevice>
    void WeightedSsePostOutputLayer<TDevice>::computeBackwardPass()
    {
        // calculate the errors
        internal::ComputeOutputErrorFn fn;
        fn.layerSize = this->size() / 2;
        fn.patTypes  = helpers::getRawPointer(this->patTypes());
        fn.targets = helpers::getRawPointer(this->_targets());
        fn.outputs = helpers::getRawPointer(this->_actualOutputs());

        int n = this->curMaxSeqLength() * this->parallelSequences() * this->size() / 2;

        thrust::transform(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(0) + n,
            this->_outputErrors().begin(),
            fn
            );
    }


    // explicit template instantiations
    template class WeightedSsePostOutputLayer<Cpu>;
    template class WeightedSsePostOutputLayer<Gpu>;

} // namespace layers
