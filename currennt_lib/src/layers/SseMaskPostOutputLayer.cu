/******************************************************************************
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "SseMaskPostOutputLayer.hpp"
#include "../helpers/getRawPointer.cuh"

#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>


namespace internal {
namespace {


    struct ComputeSseMaskFn
    {
        int layerSize;

        const char *patTypes;
        real_t *targets;
        real_t *outputs;

        __host__ __device__ real_t operator() (int index) const
        {
            // unpack the tuple
            real_t target = targets[index * 2];
            real_t actualFilter = outputs[index];
            real_t filterInput = targets[index * 2 + 1];

            // check if we have to skip this value
            int patIdx = index / layerSize;
            if (patTypes[patIdx] == PATTYPE_NONE)
                return 0;

            // calculate the error
            real_t diff = actualFilter * filterInput - target;
            return (diff * diff);
        }
    };

    struct ComputeOutputErrorFn
    {
        int layerSize;

        const char *patTypes;
        real_t *targets;
        real_t *outputs;

        __host__ __device__ real_t operator() (int index) const
        {
            // unpack the tuple
            real_t target = targets[index * 2];
            real_t actualFilter = outputs[index];
            real_t filterInput = targets[index * 2 + 1];

            // calculate the pattern index
            int patIdx = index / layerSize;

            // check if the pattern is a dummy
            if (patTypes[patIdx] == PATTYPE_NONE)
                return 0;

            // calculate the error
            real_t error = (actualFilter * filterInput - target) * filterInput;

            return error;
        }
    };
    
} // anonymous namespace
} // namespace anonymous


namespace layers {

    template <typename TDevice>
    SseMaskPostOutputLayer<TDevice>::SseMaskPostOutputLayer(const helpers::JsonValue &layerChild, Layer<TDevice> &precedingLayer)
        : PostOutputLayer<TDevice>  (layerChild, precedingLayer, precedingLayer.size() * 2)
    {
    }

    template <typename TDevice>
    SseMaskPostOutputLayer<TDevice>::~SseMaskPostOutputLayer()
    {
    }

    template <typename TDevice>
    const std::string& SseMaskPostOutputLayer<TDevice>::type() const
    {
        static const std::string s("wf");
        return s;
    }

    template <typename TDevice>
    real_t SseMaskPostOutputLayer<TDevice>::calculateError()
    {
        internal::ComputeSseMaskFn fn;
        fn.layerSize = this->size() / 2;
        fn.patTypes  = helpers::getRawPointer(this->patTypes());
        fn.targets = helpers::getRawPointer(this->_targets());
        fn.outputs = helpers::getRawPointer(this->_actualOutputs());

        int n = this->curMaxSeqLength() * this->parallelSequences() * this->size() / 2;

        real_t mse = (real_t)0.5 * thrust::transform_reduce(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(0) + n,
            fn,
            (real_t)0,
            thrust::plus<real_t>()
            );

        return mse;
    }

    template <typename TDevice>
    void SseMaskPostOutputLayer<TDevice>::computeForwardPass()
    {
    }

    template <typename TDevice>
    void SseMaskPostOutputLayer<TDevice>::computeBackwardPass()
    {
        // calculate the errors
        internal::ComputeOutputErrorFn fn;
        fn.layerSize = this->size() / 2;
        fn.patTypes  = helpers::getRawPointer(this->patTypes());
        fn.targets = helpers::getRawPointer(this->_targets());
        fn.outputs = helpers::getRawPointer(this->_actualOutputs());

        int n = this->curMaxSeqLength() * this->parallelSequences() * this->size() / 2;

        thrust::transform(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(0) + n,
            this->_outputErrors().begin(),
            fn
            );
    }


    // explicit template instantiations
    template class SseMaskPostOutputLayer<Cpu>;
    template class SseMaskPostOutputLayer<Gpu>;

} // namespace layers
